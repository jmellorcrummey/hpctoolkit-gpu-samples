
#include <hip/hip_runtime.h>
__global__
void vecAdd(float *l, float *r, float *result, size_t N) {
  size_t i = threadIdx.x;
  if (l[i] > i) {
    goto LABEL1;
  } else {
    goto LABEL2;
  }
LABEL1:
  result[i] = exp(l[i]);
  goto END;
LABEL2: 
  result[i] = l[i] + r[i];
  goto END;
END:
  return;
}
