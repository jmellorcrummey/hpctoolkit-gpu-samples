
#include <hip/hip_runtime.h>
__global__
void vecAdd(float *l, float *r, float *result, size_t N) {
  size_t i = threadIdx.x;
LABEL:
  if (l[i] > i) {
    result[i] = l[i] - r[i];
  } else {
    result[i] = l[i] + r[i];
  }
  if (i < 5) {
    ++i;
    goto LABEL;
  }
}

